#include "hip/hip_runtime.h"
// matrix vector multiplication with parallel reduction
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <chrono>
#include "helper/wtime.h"
using namespace std;


// rows = M = 10240, columns = N = 256, vector b[256].  1 block operates on 1 row. Each block has 256 threads.
// Number of blocks = 128, so increment tid by 128.
__global__ void prefix_scan (int *x_d, int n) { 
  	int tid = threadIdx.x + blockIdx.x * blockDim.x;		// initialize with block number. Tid = 0 -> 10240
 	//__shared__ has scope of block. All threads in block has access to it.
 	__shared__ int smem[8];   
	smem[threadIdx.x] = x_d[threadIdx.x];
 	//smem[tid] = x_d[tid];
 	__syncthreads(); 	//wait for all threads to copy data to smem

	while (tid < n) { 
		
	/*	if (tid == 0){
			smem[tid] = smem[tid+1];
			x_d[tid] = smem[tid];
		}
		else {
			smem[tid] += smem[tid-1];
			x_d[tid] = smem[tid];
			__syncthreads();
		}	 */
		x_d[tid] = smem[tid];
  	  
		tid += 8;	// Jump to next block which is away by 128 blocks w.r.t. current one
  	}	

} // end kernel function


int
main (int args, char **argv)
{
  int n;
  //n = 32000000;
  n = 8;	   
  int *x= (int *)malloc(sizeof(int)*n);
	
  cout << "original array: ";
  for (int i = 0; i < n; i++) {     
	  x[i] = rand () % 5 + 2;  
	  cout << x[i] << " ";       
  }   cout << endl;
	
  int *x_d;	//device storage pointers 

  hipMalloc ((void **) &x_d, sizeof (int) * n);

  hipMemcpy (x_d, x, sizeof (int) * n, hipMemcpyHostToDevice);
  
  // perform prefix_scan on GPU
  auto time_beg = wtime();  
  prefix_scan <<< 128,128 >>> (x_d,n);
	cout << "done " ;
  hipMemcpy (x, x_d, sizeof (int) * n, hipMemcpyDeviceToHost);
  auto el = wtime() - time_beg;
 // cout << "Time for <128,128> is: " << el << " Sec " << endl;

  cout << "result is: " ;
	for (int i = 0; i < n; i++){
		cout << x_d[i] << " ";
	}
	cout << endl;
	
    return 0;
}
