#include "hip/hip_runtime.h"
// matrix vector multiplication with parallel reduction
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <chrono>
#include "helper/wtime.h"
using namespace std;


__global__ void vec_mult_kernel (int *b_d, int *a_d, int n) {
int tid = threadIdx.x; // initialize with block number. Tid = 0 -> 10240
__shared__ int smem[256];
  int depth = 3;
  int d =0;
smem[threadIdx.x] = a_d[threadIdx.x];
__syncthreads(); //wait for all threads
while (tid < n) {
  if (tid == 0) { smem[0] = a_d[0]; b_d[threadIdx.x] = smem[threadIdx.x];  break;}

  while (d < depth){
      if (tid > 2^d){
        for (int k = 2^d; k<n; k++){

        smem[threadIdx.x] += a_d[threadIdx.x-1] ;
        b_d[threadIdx.x] = smem[threadIdx.x];
      __syncthreads();
          d++;
       } // for end
     } // if and
  } // end  while (int d < depth)
} // end while (tid < n)
} // end kernel function




__global__ void scanNew(int *g_odata, int *g_idata, int n)
{
 extern __shared__ float temp[]; // allocated on invocation
 int thid = threadIdx.x;
 int pout = 0, pin = 1;
 // load input into shared memory.
 // This is exclusive scan, so shift right by one and set first elt to 0
 temp[pout*n + thid] = (thid > 0) ? g_idata[thid-1] : 0;
 __syncthreads();
 for (int offset = 1; offset < n; offset *= 2)
 {
 pout = 1 - pout; // swap double buffer indices
 pin = 1 - pout;
 if (thid >= offset)
 temp[pout*n+thid] += temp[pin*n+thid - offset];
 else
 temp[pout*n+thid] = temp[pin*n+thid];
 __syncthreads();
 }
 g_odata[thid] = temp[pout*n+thid-1]; // write output
} 



int
main (int args, char **argv)
{
// configure matrix dimensions
int n = 8;
int *a= (int *)malloc(sizeof(int)*n);
int *b= (int *)malloc(sizeof(int)*n);
// Initialize matrix A and B
  cout << "array is: ";
for (int i = 0; i < n; i++) { a[i] = rand () % 5 + 2; cout << a[i] << " ";}
  cout << endl;
int *a_d, *b_d; //device storage pointers

hipMalloc ((void **) &a_d, sizeof (int) * n);
hipMalloc ((void **) &b_d, sizeof (int) * n);

hipMemcpy (a_d, a, sizeof (int) * n, hipMemcpyHostToDevice);

// perform multiplication on GPU
auto time_beg = wtime();
vec_mult_kernel <<< 128,256 >>> (b_d,a_d, n );
//scanNew <<< 128,256 >>> (b_d, a_d, n);
hipMemcpy (b, b_d, sizeof (int) * n, hipMemcpyDeviceToHost);
  cout << "result is: ";
for (int i = 0; i < n; i++) {  cout << b[i] << " ";}
  cout << endl;
auto el = wtime() - time_beg;
cout << "Time for <128,256> is: " << el << " Sec " << endl;
return 0;
}
